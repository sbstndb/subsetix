#include <stdio.h>
#include <hip/hip_runtime.h>

// Kernel CUDA pour additionner deux vecteurs
__global__ void vectorAdd(float *a, float *b, float *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}


// Kernel cuda for algebra of sets
__global__ void intersection(int* d_a_begin, int* d_a_end, int* d_b_begin, int* d_b_end){
	int i = blockIdx.x * blockDim.x + threadIdx.x;	

}


int main() {
    int n = 1000;  // Taille des vecteurs
    size_t size = n * sizeof(float);

    // Allocation des vecteurs sur l'hôte (CPU)
    float *h_a_begin = (float*)malloc(size);
    float *h_a_end = (float*)malloc(size);
    float *h_b_begin = (float*)malloc(size);
    float *h_b_end = (float*)malloc(size);
    float *h_r_begin = (float*)malloc(size);
    float *h_r_end = (float*)malloc(size);

    // Initialisation set a
    for (int i = 0; i < n; i++) {
        h_a_begin[i] = 2*i;
        h_a_end[i] = 2*i+1;
	h_b_begin[i] = 2*i ; 
	h_b_end[i] = 2*i+1 ; 
	h_r_begin[i] = 0 ; 
	h_r_end[i] = 0 ; 
    }


    // Allocation des vecteurs sur le device (GPU)
    float *d_a_begin, *d_a_end, *d_b_begin, *d_b_end, *d_r_begin, *d_r_end;
    hipMalloc(&d_a_begin, size);
    hipMalloc(&d_a_end, size);
    hipMalloc(&d_b_begin, size);
    hipMalloc(&d_b_end, size);
    hipMalloc(&d_r_begin, size);
    hipMalloc(&d_r_end, size);



    // Copie des données de l'hôte vers le device
    hipMemcpy(d_a_begin, h_a_begin, size, hipMemcpyHostToDevice);
    hipMemcpy(d_a_end, h_a_end, size, hipMemcpyHostToDevice);

    hipMemcpy(d_b_begin, h_b_begin, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b_end, h_b_end, size, hipMemcpyHostToDevice);

    hipMemcpy(d_r_begin, h_r_begin, size, hipMemcpyHostToDevice);
    hipMemcpy(d_r_end, h_r_end, size, hipMemcpyHostToDevice);


    // Configuration et lancement du kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    //vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a_begin, d_a_end, d_b_begin, d_b_end, d_r_begin, d_r_end, n);

    // Vérification des erreurs
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Erreur CUDA: %s\n", hipGetErrorString(err));
        return -1;
    }

    // Copie des résultats du device vers l'hôte
    hipMemcpy(h_r_begin, d_r_begin, size, hipMemcpyDeviceToHost);
    hipMemcpy(h_r_end, d_r_end, size, hipMemcpyDeviceToHost);

    // Affichage de quelques résultats pour vérification
    printf("Quelques résultats :\n");
    for (int i = 0; i < 5; i++) {
        printf("%.1f  %.1f  %.1f  %.1f  %.1f  %.1f \n", h_a_begin[i], h_a_end[i], h_b_begin[i], h_b_end[i], h_r_begin[i], h_r_end[i]);
    }

    // Libération de la mémoire
    free(h_a_begin);
    free(h_a_end);
    free(h_b_begin);
    free(h_b_end);
    free(h_r_begin);
    free(h_r_end);    
    hipFree(d_a_begin);
    hipFree(d_a_end);
    hipFree(d_b_begin);
    hipFree(d_b_end);
    hipFree(d_r_begin);
    hipFree(d_r_end);


    return 0;
}
