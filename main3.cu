#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void intersection_2d(
    float* d_a_begin, float* d_a_end, int a_size,
    float* d_b_begin, float* d_b_end, int b_size,
    float* d_r_begin, float* d_r_end, int* d_flags) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < a_size && j < b_size) {
        float a_begin = d_a_begin[i];
        float a_end = d_a_end[i];
        float b_begin = d_b_begin[j];
        float b_end = d_b_end[j];

        int idx = i * b_size + j;
        int overlaps = (b_begin < a_end) && (b_end > a_begin);
        d_flags[idx] = overlaps;

        if (overlaps) {
            d_r_begin[idx] = max(a_begin, b_begin);
            d_r_end[idx] = min(a_end, b_end);
        } else {
            d_r_begin[idx] = 0.0f;
            d_r_end[idx] = 0.0f;
        }
    }
}

int main() {
    int n = 10000;
    size_t size = n * sizeof(float);

    // Allocation sur l'hôte
    float *h_a_begin = (float*)malloc(size);
    float *h_a_end = (float*)malloc(size);
    float *h_b_begin = (float*)malloc(size);
    float *h_b_end = (float*)malloc(size);
    float *h_r_begin = (float*)malloc(size * n);
    float *h_r_end = (float*)malloc(size * n);
    int *h_flags = (int*)malloc(n * n * sizeof(int));

    // Initialisation des intervalles
    for (int i = 0; i < n; i++) {
        h_a_begin[i] = 4.0f * i;
        h_a_end[i] = 4.0f * i + 2.0f;
        h_b_begin[i] = 4.0f * i + 1.0f - 4.0f;
        h_b_end[i] = 4.0f * i + 3.0f - 4.0f;
        for (int j = 0; j < n; j++) {
            h_r_begin[i * n + j] = 0.0f;
            h_r_end[i * n + j] = 0.0f;
            h_flags[i * n + j] = 0;
        }
    }

    // Allocation sur le device
    float *d_a_begin, *d_a_end, *d_b_begin, *d_b_end, *d_r_begin, *d_r_end;
    int *d_flags;
    hipMalloc(&d_a_begin, size);
    hipMalloc(&d_a_end, size);
    hipMalloc(&d_b_begin, size);
    hipMalloc(&d_b_end, size);
    hipMalloc(&d_r_begin, size * n);
    hipMalloc(&d_r_end, size * n);
    hipMalloc(&d_flags, n * n * sizeof(int));

    // Copie des données vers le device
    hipMemcpy(d_a_begin, h_a_begin, size, hipMemcpyHostToDevice);
    hipMemcpy(d_a_end, h_a_end, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b_begin, h_b_begin, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b_end, h_b_end, size, hipMemcpyHostToDevice);
    hipMemcpy(d_r_begin, h_r_begin, size * n, hipMemcpyHostToDevice);
    hipMemcpy(d_r_end, h_r_end, size * n, hipMemcpyHostToDevice);
    hipMemcpy(d_flags, h_flags, n * n * sizeof(int), hipMemcpyHostToDevice);

    // Création des événements pour mesurer le temps
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Configuration de la grille 2D
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid(
        (n + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (n + threadsPerBlock.y - 1) / threadsPerBlock.y
    );

    // Enregistrement du temps de début
    hipEventRecord(start);

    // Lancement du noyau
    intersection_2d<<<blocksPerGrid, threadsPerBlock>>>(
        d_a_begin, d_a_end, n,
        d_b_begin, d_b_end, n,
        d_r_begin, d_r_end, d_flags
    );

    // Enregistrement du temps de fin
    hipEventRecord(stop);

    // Synchronisation pour attendre la fin du noyau
    hipEventSynchronize(stop);

    // Calcul du temps écoulé
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Temps d'exécution du noyau : %.3f ms\n", milliseconds);

    // Vérification des erreurs
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Erreur CUDA: %s\n", hipGetErrorString(err));
        return -1;
    }

    // Copie des résultats vers l'hôte
    hipMemcpy(h_r_begin, d_r_begin, size * n, hipMemcpyDeviceToHost);
    hipMemcpy(h_r_end, d_r_end, size * n, hipMemcpyDeviceToHost);
    hipMemcpy(h_flags, d_flags, n * n * sizeof(int), hipMemcpyDeviceToHost);

    // Affichage des résultats
    printf("Intervalles de a :\n");
    for (int i = 0; i < n; i++) {
        printf("[%.1f, %.1f[ ", h_a_begin[i], h_a_end[i]);
    }
    printf("\nIntervalles de b :\n");
    for (int i = 0; i < n; i++) {
        printf("[%.1f, %.1f[ ", h_b_begin[i], h_b_end[i]);
    }
    printf("\nIntersections :\n");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            int idx = i * n + j;
            if (h_flags[idx]) {
                printf("[%.1f, %.1f[ ", h_r_begin[idx], h_r_end[idx]);
            }
        }
    }
    printf("\n");

    // Libération des événements CUDA
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Libération de la mémoire
    free(h_a_begin); free(h_a_end); free(h_b_begin); free(h_b_end);
    free(h_r_begin); free(h_r_end); free(h_flags);
    hipFree(d_a_begin); hipFree(d_a_end); hipFree(d_b_begin); hipFree(d_b_end);
    hipFree(d_r_begin); hipFree(d_r_end); hipFree(d_flags);

    return 0;
}
